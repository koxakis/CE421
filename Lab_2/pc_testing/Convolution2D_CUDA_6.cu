#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include "gputimer.h"
#include <time.h>

unsigned int filter_radius;
GpuTimer timer;
float overal_time = 0;
clock_t start, end;
double overal_CPU_time;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter,int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			double sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = x + k;

				if (d >= 0 && d < imageW) {
					sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
				}

				h_Dst[y * imageW + x] = sum;
			}
		}
	}

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			double sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = y + k;

				if (d >= 0 && d < imageH) {
					sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
				}

				h_Dst[y * imageW + x] = sum;
			}
		}
	}

}

////////////////////////////////////////////////////////////////////////////////
// Device code
////////////////////////////////////////////////////////////////////////////////

__global__ void
convolutionRowDevice(double *d_Dst, double *d_Src, double *d_Filter,int imageW, int imageH, int filterR)
{
	int k;

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	double sum = 0;

	for (k = -filterR; k <= filterR; k++) {
		int d = row + k;

		if (d >= 0 && d < imageW) {
			//sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
			sum += d_Src[col * imageW + d] * d_Filter[filterR - k];
		}
		//h_Dst[y * imageW + x] = sum;
		d_Dst[col * imageW + row] = sum;
	}

}


__global__ void
convolutionColumnDevice(double *d_Dst, double *d_Src, double *d_Filter,int imageW, int imageH, int filterR)
{
	int k;

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	double sum = 0;

	for (k = -filterR; k <= filterR; k++) {
		int d = col + k;

		if (d >= 0 && d < imageH) {
			//sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
			sum += d_Src[d * imageW + row] * d_Filter[filterR -k];
		}
		//h_Dst[y * imageW + x] = sum;
		d_Dst[col * imageW + row] = sum;
	}

}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

	double
	*h_Filter,
	*h_Input,
	*h_Buffer,
	*h_OutputCPU,
	*h_OutputGPU;

	double
	*d_Filter,
	*d_Input,
	*d_Buffer,
	*d_OutputD;

	int imageW;
	int imageH;
	unsigned int N;
	unsigned int i;

	// Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
	// dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
	// Gia aplothta thewroume tetragwnikes eikones.

	if ( argc != 3){
		printf("Missmach in argument input \n");
		printf("1st argument: Image Size \n 2nd argument: Filter Radius \n");
		return 0;
	}

	filter_radius = atoi(argv[1]);

	N = atoi(argv[2]);
	imageH = N;
	imageW = N;

	if ( N < FILTER_LENGTH || N%2 != 0 ){
		printf ( "Wrong image size \n");
		printf ( "It should be greater than %d and a power of 2 \n", FILTER_LENGTH);
		return 0;
	}


	printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
	printf("Allocating host arrays...\n");
	// Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
	// Host mallocs

	h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
	h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
	h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
	h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
	h_OutputGPU = (double *)malloc(imageW * imageH * sizeof(double));

	if ( h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL || h_OutputGPU == NULL) {
		fprintf(stderr, "Failed to allocate Host matrices!\n");
        exit(EXIT_FAILURE);
	}

	printf("Allocating Device arrays...\n");
	// Device mallocs
	d_Filter = NULL;
	hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(double));
	cudaCheckError();

	d_Input = NULL;
	hipMalloc((void **)&d_Input, imageW * imageH * sizeof(double));
	cudaCheckError();

	d_Buffer = NULL;
	hipMalloc((void **)&d_Buffer, imageW * imageH * sizeof(double));
	cudaCheckError();

	d_OutputD = NULL;
	hipMalloc((void **)&d_OutputD, imageW * imageH * sizeof(double));
	cudaCheckError();

	// to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
	// arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
	// to convolution kai arxikopoieitai kai auth tuxaia.
	printf("Initializing Host arrays...\n");
	srand(200);

	for (i = 0; i < FILTER_LENGTH; i++) {
		h_Filter[i] = (double)(rand() % 16);
	}
	for (i = 0; i < imageW * imageH; i++) {
		h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
	}

	printf("Initializing Device arrays...\n");
	// Transfer Data to Device
	timer.Start();
	hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(double), hipMemcpyHostToDevice);
	timer.Stop();
	overal_time = overal_time + timer.Elapsed();
	cudaCheckError();

	timer.Start();
	hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(double), hipMemcpyHostToDevice);
	timer.Stop();
	overal_time = overal_time + timer.Elapsed();
	cudaCheckError();

	// To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
	printf("CPU computation...\n");

	start = clock();
	convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
	convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
	end = clock();

	// Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
	// pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

	printf("GPU computation...\n");

	// Kernel paramiters prep
	int threadsPerBlock;
	if (N >= 32){
		threadsPerBlock = 32;
	}else{
		threadsPerBlock = N;
	}
	dim3 threads(threadsPerBlock, threadsPerBlock);

	int blocksPerGrid;
	if ( N>=32){
		blocksPerGrid =  N/threads.x;
	}else{
		blocksPerGrid = 1;
	}
	dim3 grid(blocksPerGrid,blocksPerGrid);

	// convolution by rows device
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid*blocksPerGrid, threadsPerBlock*threadsPerBlock);

	timer.Start();
	convolutionRowDevice<<<grid, threads>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
	timer.Stop();
	overal_time = overal_time + timer.Elapsed();
	cudaCheckError();

	hipDeviceSynchronize();
	cudaCheckError();

	// convolution by columns device
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid*blocksPerGrid, threadsPerBlock*threadsPerBlock);

	timer.Start();
	convolutionColumnDevice<<<grid, threads>>>(d_OutputD, d_Buffer, d_Filter, imageW, imageH, filter_radius);
	timer.Stop();
	overal_time = overal_time + timer.Elapsed();
	cudaCheckError();

	hipDeviceSynchronize();
	cudaCheckError();

	// Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");

	timer.Start();
    hipMemcpy(h_OutputGPU, d_OutputD, imageW * imageH * sizeof(double), hipMemcpyDeviceToHost);
	timer.Stop();
	overal_time = overal_time + timer.Elapsed();

	cudaCheckError();

	printf("\nComparing the outputs\n");
    double max_diff=0, temp;

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
    	temp = ABS(h_OutputCPU[i] - h_OutputGPU[i]);
		if (max_diff < temp) {
			max_diff = temp;
		}

		if ( max_diff > accuracy){
			printf("The accuracy is not good enough\n" );
			break;
		}
    }

    printf("Max diff: %g\n\n", max_diff);
	printf("Time elapsed = %g ms\n", overal_time);

	overal_CPU_time = (double)(end - start) * 1000.0 / CLOCKS_PER_SEC ;
	printf ("Time elapsed on CPU = %g ms\n", overal_CPU_time);


	// free all the allocated memory
	free(h_OutputCPU);
	free(h_Buffer);
	free(h_Input);
	free(h_Filter);

	hipFree(d_OutputD);
	cudaCheckError();

	hipFree(d_Buffer);
	cudaCheckError();

	hipFree(d_Input);
	cudaCheckError();

	hipFree(d_Filter);
	cudaCheckError();

	// Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
	hipDeviceReset();


	return 0;
}
