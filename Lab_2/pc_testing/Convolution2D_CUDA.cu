/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005



////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter,int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			float sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = x + k;

				if (d >= 0 && d < imageW) {
					sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
				}

				h_Dst[y * imageW + x] = sum;
			}
		}
	}

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			float sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = y + k;

				if (d >= 0 && d < imageH) {
					sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
				}

				h_Dst[y * imageW + x] = sum;
			}
		}
	}

}

////////////////////////////////////////////////////////////////////////////////
// Device code
////////////////////////////////////////////////////////////////////////////////

__global__ void
convolutionRowDevice(float *d_Dst, float *d_Src, float *d_Filter,int imageW, int imageH, int filterR)
{
	//printf("Hello world from the convolutionRowDevice! block=%d, thread=%d\n", blockIdx.x, threadIdx.x);
	int x, y, k;
	int x_pos = blockDim.x * blockIdx.x + threadIdx.x;
	int y_pos = blockDim.y * blockIdx.y + threadIdx.y;

	printf("Row! block=%d, thread=%d, x=%d, y=%d\n", blockIdx.x, threadIdx.x, x_pos, y_pos);
	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			float sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = x + k;

				if (d >= 0 && d < imageW) {
					sum += d_Src[y * imageW + d] * d_Filter[filterR - k];
				}

				d_Dst[y * imageW + x] = sum;
			}
		}
	}
}

__global__ void
convolutionColumnDevice(float *d_Dst, float *d_Src, float *d_Filter,int imageW, int imageH, int filterR)
{
	int x, y, k;
	int x_pos = blockDim.x * blockIdx.x + threadIdx.x;
	int y_pos = blockDim.y * blockIdx.y + threadIdx.y;

	printf("Column! block=%d, thread=%d, x=%d, y=%d\n", blockIdx.x, threadIdx.x, x_pos, y_pos);

	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			float sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = y + k;

				if (d >= 0 && d < imageH) {
					sum += d_Src[d * imageW + x] * d_Filter[filterR - k];
				}

				d_Dst[y * imageW + x] = sum;
			}
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

	float
	*h_Filter,
	*h_Input,
	*h_Buffer,
	*h_OutputCPU,
	*h_OutputGPU;

	float
	*d_Filter,
	*d_Input,
	*d_Buffer,
	*d_OutputD;

	int imageW;
	int imageH;
	unsigned int N;
	unsigned int i;

	hipError_t err = hipSuccess;

	// Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
	// dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
	// Gia aplothta thewroume tetragwnikes eikones.

	if ( argc != 3){
		printf("Missmach in argument input \n");
		return 0;
	}

	filter_radius = atoi(argv[1]);

	N = atoi(argv[2]);
	imageH = N;
	imageW = N;

	if ( N < FILTER_LENGTH || N%2 != 0 ){
		printf ( "Wrong image size \n");
		printf ( "It should be greater than %d and a power of 2 \n", FILTER_LENGTH);
		return 0;
	}


	printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
	printf("Allocating and initializing host arrays...\n");
	// Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
	// Host mallocs
	h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
	h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
	h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
	h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
	h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));

	if ( h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL || h_OutputGPU == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
	}

	// Device mallocs
	d_Filter = NULL;
	err = hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device Filter (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	d_Input = NULL;
	err = hipMalloc((void **)&d_Input, imageW * imageH * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device Input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	d_Buffer = NULL;
	err = hipMalloc((void **)&d_Buffer, imageW * imageH * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	d_OutputD = NULL;
	err = hipMalloc((void **)&d_OutputD, imageW * imageH * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
	// arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
	// to convolution kai arxikopoieitai kai auth tuxaia.

	srand(200);

	for (i = 0; i < FILTER_LENGTH; i++) {
		h_Filter[i] = (float)(rand() % 16);
	}
	for (i = 0; i < imageW * imageH; i++) {
		h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
	}

	// Transfer Data to Device
	err = hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
	printf("CPU computation...\n");

	convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
	convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles


	// Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
	// pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

	printf("GPU computation...\n");
	// Error code to check return values for CUDA calls

	// Kernel paramiters prep
	int threadsPerBlock = 1024;
	int blocksPerGrid = 1;
	//int blocksPerGrid =(imageH + threadsPerBlock - 1) / threadsPerBlock;

	// convolution by rows device
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	convolutionRowDevice<<<blocksPerGrid, threadsPerBlock>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
	err = hipGetLastError();

	hipDeviceSynchronize();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch convolutionRowDevice kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// convolution by columns device
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	convolutionColumnDevice<<<blocksPerGrid, threadsPerBlock>>>(d_OutputD, d_Buffer, d_Filter, imageW, imageH, filter_radius);
	err = hipGetLastError();

	hipDeviceSynchronize();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch convolutionColumnDevice kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_OutputGPU, d_OutputD, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy result from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	printf(" Comparing the outputs\n");
    double sum = 0, delta = 0;

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
        delta += (h_OutputGPU[i] - h_OutputCPU[i]) * (h_OutputGPU[i] - h_OutputCPU[i]);
        sum   += h_OutputCPU[i] * h_OutputCPU[i];
    }
	double L2norm = sqrt(delta / sum);
    printf(" Relative L2 norm: %E\n\n", L2norm);


	// free all the allocated memory
	free(h_OutputCPU);
	free(h_Buffer);
	free(h_Input);
	free(h_Filter);

	// Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
	hipDeviceReset();


	return 0;
}
