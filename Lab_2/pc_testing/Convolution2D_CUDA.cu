/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005



////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter,
	int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			float sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = x + k;

				if (d >= 0 && d < imageW) {
					sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
				}

				h_Dst[y * imageW + x] = sum;
			}
		}
	}

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
	int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			float sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = y + k;

				if (d >= 0 && d < imageH) {
					sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
				}

				h_Dst[y * imageW + x] = sum;
			}
		}
	}

}

////////////////////////////////////////////////////////////////////////////////
// Device code
////////////////////////////////////////////////////////////////////////////////

__global__ void
convolutionDevice()
{
	//printf("Hello world from the device! block=%d, thread=%d\n", blockIdx.x, threadIdx.x);
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

	float
	*h_Filter,
	*h_Input,
	*h_Buffer,
	*h_OutputCPU;

	int imageW;
	int imageH;
	unsigned int N;
	unsigned int i;

	//printf("Enter filter radius : ");
	//scanf("%d", &filter_radius);

	// Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
	// dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
	// Gia aplothta thewroume tetragwnikes eikones.

	//printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
	//scanf("%d", &imageW);

	if ( argc != 3){
		printf("Missmach in argument input \n");
		return 0;
	}

	filter_radius = atoi(argv[1]);

	N = atoi(argv[2]);
	imageH = N;
	imageW = N;

	if ( N < FILTER_LENGTH || N%2 != 0 ){
		printf ( "Wrong image size \n");
		printf ( "It should be greater than %d and a power of 2 \n", FILTER_LENGTH);
		return 0;
	}


	printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
	printf("Allocating and initializing host arrays...\n");
	// Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
	h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
	h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
	h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
	h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));

	if ( h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
	}

	// to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
	// arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
	// to convolution kai arxikopoieitai kai auth tuxaia.

	srand(200);

	for (i = 0; i < FILTER_LENGTH; i++) {
		h_Filter[i] = (float)(rand() % 16);
	}

	for (i = 0; i < imageW * imageH; i++) {
		h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
	}


	// To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
	printf("CPU computation...\n");

	convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
	convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles


	// Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
	// pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

	printf("GPU computation...\n");
	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

	//start the kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =(imageH + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	convolutionDevice<<<blocksPerGrid, threadsPerBlock>>>();
	err = hipGetLastError();

	hipDeviceSynchronize();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch convolutionDevice kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// free all the allocated memory
	free(h_OutputCPU);
	free(h_Buffer);
	free(h_Input);
	free(h_Filter);

	// Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
	hipDeviceReset();


	return 0;
}
