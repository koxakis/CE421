#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include "gputimer.h"
#include <time.h>

unsigned int filter_radius;
GpuTimer timer;
double overal_time = 0;
clock_t start, end;
double overal_CPU_time;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005

// FLOAT_D for floats DOUBLE_D for doubles
// Remove to use integer data type
#define DOUBLE_D

// Use 48KB for shared memory and 16KB for L1 cache
// Remove for opposite
//#define PREF_SHARED

// Variable data types
#ifdef FLOAT_D
typedef float vart_t;
#elif defined DOUBLE_D
typedef double vart_t;
#else
typedef int vart_t;
#endif


#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(vart_t *h_Dst, vart_t *h_Src, vart_t *h_Filter,int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			vart_t sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = x + k;

				if (d >= 0 && d < imageW) {
					sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
				}

				h_Dst[y * imageW + x] = sum;
			}
		}
	}

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(vart_t *h_Dst, vart_t *h_Src, vart_t *h_Filter,int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			vart_t sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = y + k;

				if (d >= 0 && d < imageH) {
					sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
				}

				h_Dst[y * imageW + x] = sum;
			}
		}
	}

}

////////////////////////////////////////////////////////////////////////////////
// Device code
////////////////////////////////////////////////////////////////////////////////

__global__ void
convolutionRowDevice(vart_t *d_Dst, vart_t *d_Src, vart_t *d_Filter,int imageW, int imageH, int filterR)
{
	int k;

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	vart_t sum = 0;

	for (k = -filterR; k <= filterR; k++) {
		int d = row + k;

		if (d >= 0 && d < imageW) {
			//sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
			sum += d_Src[col * imageW + d] * d_Filter[filterR - k];
		}
		//h_Dst[y * imageW + x] = sum;
		d_Dst[col * imageW + row] = sum;
	}

}


__global__ void
convolutionColumnDevice(vart_t *d_Dst, vart_t *d_Src, vart_t *d_Filter,int imageW, int imageH, int filterR)
{
	int k;

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	vart_t sum = 0;

	for (k = -filterR; k <= filterR; k++) {
		int d = col + k;

		if (d >= 0 && d < imageH) {
			//sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
			sum += d_Src[d * imageW + row] * d_Filter[filterR -k];
		}
		//h_Dst[y * imageW + x] = sum;
		d_Dst[col * imageW + row] = sum;
	}

}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

	vart_t
	*h_Filter,
	*h_Input,
	*h_Buffer,
	*h_OutputCPU,
	*h_OutputGPU;

	vart_t
	*d_Filter,
	*d_Input,
	*d_Buffer,
	*d_OutputD;

	int imageW;
	int imageH;
	unsigned int N;
	unsigned int i;

	// Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
	// dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
	// Gia aplothta thewroume tetragwnikes eikones.

	if ( argc != 3){
		printf("Missmach in argument input \n");
		printf("1st argument: Image Size \n 2nd argument: Filter Radius \n");
		return 0;
	}

	filter_radius = atoi(argv[1]);

	N = atoi(argv[2]);
	imageH = N;
	imageW = N;

	if ( N < FILTER_LENGTH || N%2 != 0 ){
		printf ( "Wrong image size \n");
		printf ( "It should be greater than %d and a power of 2 \n", FILTER_LENGTH);
		return 0;
	}


	printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
	printf("Allocating host arrays...\n");
	// Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
	// Host mallocs

	h_Filter    = (vart_t *)malloc(FILTER_LENGTH * sizeof(vart_t));
	h_Input     = (vart_t *)malloc(imageW * imageH * sizeof(vart_t));
	h_Buffer    = (vart_t *)malloc(imageW * imageH * sizeof(vart_t));
	h_OutputCPU = (vart_t *)malloc(imageW * imageH * sizeof(vart_t));
	h_OutputGPU = (vart_t *)malloc(imageW * imageH * sizeof(vart_t));

	if ( h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL || h_OutputGPU == NULL) {
		fprintf(stderr, "Failed to allocate Host matrices!\n");
        exit(EXIT_FAILURE);
	}

	printf("Allocating Device arrays...\n");
	// Device mallocs
	d_Filter = NULL;
	hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(vart_t));
	cudaCheckError();

	d_Input = NULL;
	hipMalloc((void **)&d_Input, imageW * imageH * sizeof(vart_t));
	cudaCheckError();

	d_Buffer = NULL;
	hipMalloc((void **)&d_Buffer, imageW * imageH * sizeof(vart_t));
	cudaCheckError();

	d_OutputD = NULL;
	hipMalloc((void **)&d_OutputD, imageW * imageH * sizeof(vart_t));
	cudaCheckError();

	// to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
	// arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
	// to convolution kai arxikopoieitai kai auth tuxaia.
	printf("Initializing Host arrays...\n");
	srand(200);

	for (i = 0; i < FILTER_LENGTH; i++) {
		h_Filter[i] = (vart_t)(rand() % 16);
	}
	for (i = 0; i < imageW * imageH; i++) {
		h_Input[i] = (vart_t)rand() / ((vart_t)RAND_MAX / 255) + (vart_t)rand() / (vart_t)RAND_MAX;
	}

	printf("Initializing Device arrays...\n");
	// Transfer Data to Device
	timer.Start();
	hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(vart_t), hipMemcpyHostToDevice);
	timer.Stop();
	overal_time = overal_time + timer.Elapsed();
	cudaCheckError();

	timer.Start();
	hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(vart_t), hipMemcpyHostToDevice);
	timer.Stop();
	overal_time = overal_time + timer.Elapsed();
	cudaCheckError();

	// To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
	printf("CPU computation...\n");

	start = clock();
	convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
	convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
	end = clock();

	// Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
	// pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

	printf("GPU computation...\n");

	// Kernel paramiters prep

	#ifdef PREF_SHARED
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(convolutionRowDevice), hipFuncCachePreferShared);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(convolutionColumnDevice), hipFuncCachePreferShared);
	#else
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(convolutionRowDevice), hipFuncCachePreferL1);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(convolutionColumnDevice), hipFuncCachePreferL1);
	#endif

	int threadsPerBlock;
	if (N >= 32){
		threadsPerBlock = 32;
	}else{
		threadsPerBlock = N;
	}
	dim3 threads(threadsPerBlock, threadsPerBlock);

	int blocksPerGrid;
	if ( N>=32){
		blocksPerGrid =  N/threads.x;
	}else{
		blocksPerGrid = 1;
	}
	dim3 grid(blocksPerGrid,blocksPerGrid);

	// convolution by rows device
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid*blocksPerGrid, threadsPerBlock*threadsPerBlock);

	timer.Start();
	convolutionRowDevice<<<grid, threads>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
	timer.Stop();
	overal_time = overal_time + timer.Elapsed();
	cudaCheckError();

	hipDeviceSynchronize();
	cudaCheckError();

	// convolution by columns device
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid*blocksPerGrid, threadsPerBlock*threadsPerBlock);

	timer.Start();
	convolutionColumnDevice<<<grid, threads>>>(d_OutputD, d_Buffer, d_Filter, imageW, imageH, filter_radius);
	timer.Stop();
	overal_time = overal_time + timer.Elapsed();
	cudaCheckError();

	hipDeviceSynchronize();
	cudaCheckError();

	// Copy the device result vector in device memory to the host result vector
    // in host memorycomment
    printf("Copy output data from the CUDA device to the host memory\n");

	timer.Start();
    hipMemcpy(h_OutputGPU, d_OutputD, imageW * imageH * sizeof(vart_t), hipMemcpyDeviceToHost);
	timer.Stop();
	overal_time = overal_time + timer.Elapsed();

	cudaCheckError();

	printf("\nComparing the outputs\n");
    vart_t max_diff=0, temp;

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
    	temp = ABS(h_OutputCPU[i] - h_OutputGPU[i]);
		if (max_diff < temp) {
			max_diff = temp;
		}

		if ( max_diff > accuracy){
			printf("The accuracy is not good enough\n" );
			break;
		}
    }

    printf("Max diff: %g\n\n", max_diff);
	printf("Time elapsed on GPU = %g ms\n", overal_time);

	overal_CPU_time = (double)(end - start) * 1000.0 / CLOCKS_PER_SEC ;
	printf ("Time elapsed on CPU = %g ms\n", overal_CPU_time);


	// free all the allocated memory
	free(h_OutputCPU);
	free(h_Buffer);
	free(h_Input);
	free(h_Filter);

	hipFree(d_OutputD);
	cudaCheckError();

	hipFree(d_Buffer);
	cudaCheckError();

	hipFree(d_Input);
	cudaCheckError();

	hipFree(d_Filter);
	cudaCheckError();

	// Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
	hipDeviceReset();


	return 0;
}
