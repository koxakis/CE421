#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include "gputimer.h"
#include <time.h>

unsigned int filter_radius;
GpuTimer timer;
double overal_GPU_time = 0, overal_data_transfer_time = 0;
clock_t start, end;
double overal_CPU_time;

int threadsPerBlock;
int blocksPerGrid;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005

// Filter array size change accordingly
#define FILTER_ARRAY_SIZE 3
// Change the size of the tile and GPU thread block
#define TILE_WIDTH 4
#define THREADS_PER_BLOCK 4

// Remove to set block and tile size indepentently
//#define LOCK_BLOCK_TILE

// Use 48KB for shared memory and 16KB for L1 cache
// Remove for opposite
#define PREF_SHARED

// Enable to compare with the CPU result
#define DEBUG

// FLOAT_D for floats DOUBLE_D for doubles
// Remove to use integer data type
//#define FLOAT_D

// Variable data types
#ifdef FLOAT_D
typedef float vart_t;
#elif defined DOUBLE_D
typedef double vart_t;
#else
typedef int vart_t;
#endif

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(vart_t *h_Dst, vart_t *h_Src, vart_t *h_Filter,int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			vart_t sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = x + k;

				if (d >= 0 && d < imageW) {
					sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
				}
				h_Dst[y * imageW + x] = sum;
			}
		}
	}

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(vart_t *h_Dst, vart_t *h_Src, vart_t *h_Filter,int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			vart_t sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = y + k;

				if (d >= 0 && d < imageH) {
					sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
				}
				h_Dst[y * imageW + x] = sum;
			}
			//printf("CPU %d %d\n", h_Dst[y * imageW + x], y * imageW + x);
		}
	}

}

////////////////////////////////////////////////////////////////////////////////
// Device code
////////////////////////////////////////////////////////////////////////////////

__device__ __constant__ vart_t d_Filter[FILTER_ARRAY_SIZE];

__global__ void
convolutionRowDevice(vart_t *d_Dst, vart_t *d_Src, int imageW, int imageH, int filterR, int blk_per_grid)
{
	int k;

	// Thread location in the grid
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	//int row = blockIdx.y * blockDim.y + threadIdx.y;

	int generic_loc = col + (threadIdx.y * imageW) + (blockIdx.y * blockDim.y) * imageW;
	
	// Shared memory decleration
	__shared__ vart_t tiled_block[TILE_WIDTH + FILTER_ARRAY_SIZE-1][ TILE_WIDTH ];

	vart_t sum = 0;

	// Collaboratively load tiles into __shared__
	// Left load
	if ((col - filterR) < 0){
		tiled_block[threadIdx.x][threadIdx.y] = 0;
	}else{
		tiled_block[threadIdx.x][threadIdx.y] = d_Src[generic_loc - filterR ];
	}
	// Right load
	if ((col + filterR) > imageW-1) {
		tiled_block[threadIdx.x + blockDim.x][threadIdx.y] = 0;
	}else{
		tiled_block[threadIdx.x + blockDim.x][threadIdx.y] = d_Src[generic_loc + filterR];
	}

	__syncthreads();

	for (k = -filterR; k <= filterR; k++) {
		//int d = threadIdx.x + k;

		//if (d >= 0 && d < TILE_WIDTH) {
			//sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
		sum += tiled_block[(threadIdx.x+filterR) + k][threadIdx.y] * d_Filter[filterR + k];
		//}
		//h_Dst[y * imageW + x] = sum;

		//d_Dst[generic_loc] = sum;
	}
	d_Dst[generic_loc] = sum;
	//printf(" %d %d\n", d_Dst[col * imageW + row], col * imageW + row);
}


__global__ void
convolutionColumnDevice(vart_t *d_Dst, vart_t *d_Src, int imageW, int imageH, int filterR, int blk_per_grid)
{
	int k;

	// Thread location in the grid
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	int generic_loc = col + (threadIdx.y * imageW) + (blockIdx.y * blockDim.y) * imageW;

	// Shared memory decleration
	__shared__ vart_t tiled_block[TILE_WIDTH][ TILE_WIDTH + (FILTER_ARRAY_SIZE-1) ];

	vart_t sum = 0;

	// Collaboratively load tiles into __shared__
	// Main load
	if ((row - filterR) < 0){
		tiled_block[threadIdx.x][threadIdx.y] = 0;
	}else{
		tiled_block[threadIdx.x][threadIdx.y] = d_Src[generic_loc - (imageW * filterR)];
	}
	// Lower load
	if ((row + filterR) > imageW-1) {
		tiled_block[threadIdx.x][threadIdx.y + blockDim.y] = 0;
	}else{
		tiled_block[threadIdx.x][threadIdx.y + blockDim.y] = d_Src[generic_loc + (imageW * filterR) ];
	}

	__syncthreads();

	//for (k = -filterR; k <= filterR; k++) {
	for (k = -filterR; k <= filterR; k++) {
		//int d = threadIdx.y + k;

		//if (d >= 0 && d < TILE_WIDTH) {
			//sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
			sum += tiled_block[threadIdx.x][(filterR + threadIdx.y) + k] * d_Filter[filterR + k];
		//}
		//h_Dst[y * imageW + x] = sum;

		//d_Dst[generic_loc] = sum;
	}
	d_Dst[generic_loc] = sum;
	//printf(" %d %d\n", d_Dst[col * imageW + row], col * imageW + row);

}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

	vart_t
	h_Filter[FILTER_ARRAY_SIZE],
	*h_Input,
	*h_Buffer,
	*h_OutputCPU,
	*h_OutputGPU;

	vart_t
	*d_Input,
	*d_Buffer,
	*d_OutputD;

	int imageW;
	int imageH;
	unsigned int N;
	unsigned int i;

	// Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
	// dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
	// Gia aplothta thewroume tetragwnikes eikones.

	if ( argc != 3){
		printf("Missmach in argument input \n");
		printf("1st argument: Image Size \n 2nd argument: Filter Radius \n");
		return 0;
	}

	filter_radius = atoi(argv[1]);

	N = atoi(argv[2]);
	imageH = N;
	imageW = N;

	if ( N < FILTER_LENGTH || N%2 != 0 ){
		printf ( "Wrong image size \n");
		printf ( "It should be greater than %d and a power of 2 \n", FILTER_LENGTH);
		return 0;
	}


	printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
	printf("Allocating host arrays...\n");
	// Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
	// Host mallocs

	h_Input     = (vart_t *)malloc(imageW * imageH * sizeof(vart_t));
	h_Buffer    = (vart_t *)malloc(imageW * imageH * sizeof(vart_t));
	h_OutputCPU = (vart_t *)malloc(imageW * imageH * sizeof(vart_t));
	h_OutputGPU = (vart_t *)malloc(imageW * imageH * sizeof(vart_t));

	if ( h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL || h_OutputGPU == NULL) {
		fprintf(stderr, "Failed to allocate Host arrays!\n");
        exit(EXIT_FAILURE);
	}

	printf("Allocating Device arrays...\n");

	// Device mallocs
	d_Input = NULL;
	hipMalloc((void **)&d_Input, imageW * imageH * sizeof(vart_t));
	cudaCheckError();

	d_Buffer = NULL;
	hipMalloc((void **)&d_Buffer, imageW * imageH * sizeof(vart_t));
	cudaCheckError();

	d_OutputD = NULL;
	hipMalloc((void **)&d_OutputD, imageW * imageH * sizeof(vart_t));
	cudaCheckError();

	// to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
	// arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
	// to convolution kai arxikopoieitai kai auth tuxaia.
	printf("Initializing Host arrays...\n");
	srand(200);

	// Host arrays initialization
	for (i = 0; i < FILTER_LENGTH; i++) {
		h_Filter[i] = (vart_t)(rand() % 16);
	}
	for (int i = 0; i < imageW * imageH; i++) {
		h_Input[i] = (vart_t)rand() / ((vart_t)RAND_MAX / 255) + (vart_t)rand() / (vart_t)RAND_MAX;
	}
	/*
	for (int i = 0; i < FILTER_LENGTH; i++) {
		printf("%d\n", h_Filter[i]);
	}
	*//*
	for (int i = 0; i < imageH * imageW; i++) {
		printf(" %d \n", h_Input[i]);
	} */
	printf("Initializing Device arrays...\n");
	// Transfer Data to Device
	timer.Start();
	hipMemcpyToSymbol(HIP_SYMBOL( d_Filter), h_Filter, FILTER_ARRAY_SIZE * sizeof(vart_t));
	timer.Stop();
	overal_data_transfer_time += timer.Elapsed();
	cudaCheckError();

	timer.Start();
	hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(vart_t), hipMemcpyHostToDevice);
	timer.Stop();
	overal_data_transfer_time += timer.Elapsed();
	cudaCheckError();

	// To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
	printf("CPU computation...\n");

#ifdef DEBUG
	// CPU execution
	start = clock();
	convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
	convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
	end = clock();
#endif
	// Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
	// pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

	printf("GPU computation...\n");

	// Kernel paramiters prep
#ifdef LOCK_BLOCK_TILE
	threadsPerBlock = TILE_WIDTH;
	blocksPerGrid = N / TILE_WIDTH;
#else
	if (N >= 32){
		threadsPerBlock = THREADS_PER_BLOCK;
		blocksPerGrid =  N/THREADS_PER_BLOCK;
	}else{
		threadsPerBlock = N;
		blocksPerGrid = 1;
	}
#endif

	dim3 threads(threadsPerBlock, threadsPerBlock);
	dim3 grid(blocksPerGrid,blocksPerGrid);

#ifdef PREF_SHARED
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(convolutionRowDevice), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(convolutionColumnDevice), hipFuncCachePreferShared);
#else
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(convolutionRowDevice), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(convolutionColumnDevice), hipFuncCachePreferL1);
#endif

	// convolution by rows device
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid*blocksPerGrid, threadsPerBlock*threadsPerBlock);

	timer.Start();
	convolutionRowDevice<<<grid, threads>>>(d_Buffer, d_Input, imageW, imageH, filter_radius, blocksPerGrid);
	timer.Stop();
	overal_GPU_time += timer.Elapsed();
	cudaCheckError();

	hipDeviceSynchronize();
	cudaCheckError();

	// convolution by columns device
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid*blocksPerGrid, threadsPerBlock*threadsPerBlock);

	timer.Start();
	convolutionColumnDevice<<<grid, threads>>>(d_OutputD, d_Buffer, imageW, imageH, filter_radius, blocksPerGrid);
	timer.Stop();
	overal_GPU_time += timer.Elapsed();
	cudaCheckError();

	hipDeviceSynchronize();
	cudaCheckError();

	// Copy the device result vector in device memory to the host result vector
    // in host memorycomment
    printf("Copy output data from the CUDA device to the host memory\n");

	timer.Start();
    hipMemcpy(h_OutputGPU, d_OutputD, imageW * imageH * sizeof(vart_t), hipMemcpyDeviceToHost);
	timer.Stop();
	overal_data_transfer_time += timer.Elapsed();

	cudaCheckError();
#ifdef DEBUG
	printf("\nComparing the outputs\n");

    vart_t max_diff=0, temp;

    for (int i = 0; i < imageW * imageH; i++)
    {
    	temp = ABS(h_OutputCPU[i] - h_OutputGPU[i]);
		if (temp > 0) {
			printf("pos i %d CPU %d GPU %d\n", i, h_OutputCPU[i], h_OutputGPU[i]);
		}
		if (max_diff < temp) {
			max_diff = temp;
		}
/*
		if ( max_diff > accuracy){
			printf("The accuracy is not good enough\n" );
			break;
		}
		*/
    }
#ifdef FLOAT_D
    printf("Max diff: %g \n\n", max_diff);
#else
	printf("Max diff: %d \n\n", max_diff);
#endif

	overal_CPU_time = (double)(end - start) * 1000.0 / CLOCKS_PER_SEC ;
	printf ("Time elapsed on CPU = %g ms\n", overal_CPU_time);

	printf("\nSettings used: \nTile width %d block size %d ", TILE_WIDTH, threads.x);
	printf("\nTime elapsed on GPU( computation) = %g ms\n", overal_GPU_time);

	printf("\nTime elapsed on GPU( memory transfers) = %g ms", overal_data_transfer_time);

	printf("\nTime elapsed on GPU( overal) = %g ms\n", overal_GPU_time + overal_data_transfer_time);

#else

	printf("\nSettings used: \nTile width %d block size %d ", TILE_WIDTH, threads.x);
	printf("Time elapsed on GPU( computation) = %g ms\n", overal_GPU_time);
#endif

	// free all the allocated memory
	free(h_OutputCPU);
	free(h_Buffer);
	free(h_Input);

	hipFree(d_OutputD);
	cudaCheckError();

	hipFree(d_Buffer);
	cudaCheckError();

	hipFree(d_Input);
	cudaCheckError();

	// Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
	hipDeviceReset();


	return 0;
}
